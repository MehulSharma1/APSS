
//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <algorithm>
#include <vector>
//#include <omp.h>
//#include <hip/hip_runtime.h>
using namespace std;
	
vector < vector <int> > V;
vector < pair<int, vector<int>> > I;
const int N = 10;

//Comparator Function for Bayardo Omega Permutation
bool bomega(vector<int> f, vector<int> l)
{ return (*max_element(f.begin(), f.end()) < *max_element(l.begin(), l.end())); }
void bayardo_omega()
{
	/* Sorting Vectors in V based on 
	   Maximum Value in each Vector */
	sort(V.begin(), V.end(), bomega);
}

bool ssize(int f)
{ return (f > 0); }
bool somega(vector<int> f, vector<int> l)
{ return (count_if(f.begin(), f.end(), ssize) < count_if(l.begin, l.end, ssize)); }
void sarawagi_omega()
{
	/* Sorting Vectors in V based on 
	   Number of Non-Zero components 
	   in each Vector */
	sort(V.begin(), V.end(), somega);
}

//Comparator Function for Pi Permutation
bool bpi(pair<int, vector<int>> f, pair<int, vector<int>> l)
{ return (f.second.size() < l.second.size()); }
void bayardo_pi()
{
	/* Sort Inverted Index Map I based
	   on Dimension Density */
	sort(I.begin(), I.end(), bpi);
}

void preprocess()
{
	/* Assume a list of Vectors : V[]
	   in an N-Dimensional Space, we 
	   build an Inverted Index Map I,
	   which contains N pair of 
	   elements, where the first element
	   is the dimension, and the second
	   is a vector which contain indices
	   of the vectors that have a 
	   projection in that dimension. */

	vector<int> v[N];
	int i, j;
	for (i = 0; i < V.size(); ++i)
	{
		//#pragma omp parallel for shared(i,V,I)			//All the Dimensions can be checked for in Parallel
		for (j = 0; j < N; ++j)
			if (V[i][j] > 0)
				v[j].push_back(i);

		I.push_back(make_pair(j, v[j]));
	}

	bayardo_omega();
	//sarawagi_omega();

	bayardo_pi();
}